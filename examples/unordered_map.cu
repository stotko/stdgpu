#include "hip/hip_runtime.h"
/*
 *  Copyright 2019 Patrick Stotko
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdgpu/unordered_map.cuh>

namespace stdgpu
{

template <>
struct hash<longlong3>
{
    inline STDGPU_HOST_DEVICE std::size_t
    operator()(const longlong3 &key) const
    {
        return key.x * 73856093 ^ key.y * 19349669 ^ key.z * 83492791;
    }
};

} // namespace stdgpu

STDGPU_HOST_DEVICE bool
operator==(const longlong3 &lhs, const longlong3 &rhs)
{
    return lhs.x == rhs.x && lhs.y == rhs.y && lhs.z == rhs.z;
};

using voxel_map = stdgpu::unordered_map<longlong3, float>;

__global__ void
add_keys_and_values(const stdgpu::index_t n,
                    const longlong3* voxel_idx,
                    const float* voxel_tsd,
                    voxel_map map)
{
    stdgpu::index_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n)
        return;

    longlong3 idx = voxel_idx[i];
    float tsd = voxel_tsd[i];

    map.insert(thrust::make_pair(idx, tsd));
}

int main()
{
    stdgpu::index_t n = 3;

    longlong3 voxel_idx_host[n] = {
        {0, 0, 0},
        {0, 0, 1},
        {0, 1, 0}};
    longlong3* voxel_idx;
    hipMalloc(&voxel_idx, sizeof(longlong3)*n);
    hipMemcpy(voxel_idx, voxel_idx_host, sizeof(longlong3)*n, hipMemcpyHostToDevice); 

    float voxel_tsd_host[n] = {0.1, 0.002, -0.3};
    float* voxel_tsd;
    hipMalloc(&voxel_tsd, sizeof(float)*n);
    hipMemcpy(voxel_tsd, voxel_tsd_host, sizeof(float)*n, hipMemcpyHostToDevice); 

    voxel_map map = voxel_map::createDeviceObject(1024, n);

    stdgpu::index_t threads = 128;
    stdgpu::index_t blocks = (n + threads - 1) / threads;
    add_keys_and_values<<<blocks, threads>>>(n, voxel_idx, voxel_tsd, map);
    hipDeviceSynchronize();

    std::cout << "Number of elements is " << map.size() << " (" << 3 << " expected)" << std::endl;

    hipFree(voxel_idx);
    hipFree(voxel_tsd);

    
    n = 1;

    longlong3 additional_voxel_idx_host[n] = {{0, 0, 0}};
    hipMalloc(&voxel_idx, sizeof(longlong3)*n);
    hipMemcpy(voxel_idx, additional_voxel_idx_host, sizeof(longlong3)*n, hipMemcpyHostToDevice); 

    float additional_voxel_tsd_host[n] = {0.0};
    hipMalloc(&voxel_tsd, sizeof(float)*n);
    hipMemcpy(voxel_tsd, additional_voxel_tsd_host, sizeof(float)*n, hipMemcpyHostToDevice); 

    threads = 128;
    blocks = (n + threads - 1) / threads;
    add_keys_and_values<<<blocks, threads>>>(n, voxel_idx, voxel_tsd, map);

    std::cout << "Number of elements after overwriting an existing element with a different value is " << map.size() << " (" << 3 << " expected)" << std::endl;

    hipFree(voxel_idx);
    hipFree(voxel_tsd);


    voxel_map::destroyDeviceObject(map);
}
