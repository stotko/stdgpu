#include "hip/hip_runtime.h"
/*
 *  Copyright 2019 Patrick Stotko
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdgpu/atomic.inc>


#include <gtest/gtest.h>

#include <cmath>
#include <limits>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include <test_utils.h>
#include <stdgpu/atomic.cuh>
#include <stdgpu/iterator.h>
#include <stdgpu/memory.h>



class stdgpu_cuda_atomic : public ::testing::Test
{
    protected:
        // Called before each test
        virtual void SetUp()
        {

        }

        // Called after each test
        virtual void TearDown()
        {

        }
};



template <typename T>
class subtract
{
    public:
        subtract(T* value)
            : _value(value)
        {

        }

        STDGPU_DEVICE_ONLY void
        operator()(const T x)
        {
            atomicSub(_value, x);
        }

    private:
        T* _value;
};


TEST_F(stdgpu_cuda_atomic, unsigned_long_long_int_sub)
{
    const stdgpu::index64_t N = 10000000;

    // Create sequence
    unsigned long long int* numbers = createDeviceArray<unsigned long long int>(N);

    thrust::sequence(stdgpu::device_begin(numbers), stdgpu::device_end(numbers),
                     static_cast<unsigned long long int>(1));

    unsigned long long int* sum = createDeviceArray<unsigned long long int>(1, N * (N + 1) / 2);


    thrust::for_each(stdgpu::device_begin(numbers), stdgpu::device_end(numbers),
                     subtract<unsigned long long int>(sum));


    unsigned long long int* host_sum = copyCreateDevice2HostArray(sum, 1);

    EXPECT_EQ(*host_sum, static_cast<unsigned long long int>(0));


    destroyDeviceArray<unsigned long long int>(numbers);
    destroyDeviceArray<unsigned long long int>(sum);
    destroyHostArray<unsigned long long int>(host_sum);
}


TEST_F(stdgpu_cuda_atomic, unsigned_long_long_int_sub_zero_pattern)
{
    unsigned long long int* number  = createDeviceArray<unsigned long long int>(1, 0);  // zero pattern
    unsigned long long int* sum     = createDeviceArray<unsigned long long int>(1, 42);


    thrust::for_each(stdgpu::device_begin(number), stdgpu::device_end(number),
                     subtract<unsigned long long int>(sum));


    unsigned long long int* host_sum = copyCreateDevice2HostArray(sum, 1);

    EXPECT_EQ(*host_sum, static_cast<unsigned long long int>(42));


    destroyDeviceArray<unsigned long long int>(number);
    destroyDeviceArray<unsigned long long int>(sum);
    destroyHostArray<unsigned long long int>(host_sum);
}


TEST_F(stdgpu_cuda_atomic, unsigned_long_long_int_sub_one_pattern)
{
    unsigned long long int* number  = createDeviceArray<unsigned long long int>(1, std::numeric_limits<unsigned long long int>::max()); // one pattern
    unsigned long long int* sum     = createDeviceArray<unsigned long long int>(1, 42);


    thrust::for_each(stdgpu::device_begin(number), stdgpu::device_end(number),
                     subtract<unsigned long long int>(sum));


    unsigned long long int* host_sum = copyCreateDevice2HostArray(sum, 1);

    EXPECT_EQ(*host_sum, static_cast<unsigned long long int>(43));


    destroyDeviceArray<unsigned long long int>(number);
    destroyDeviceArray<unsigned long long int>(sum);
    destroyHostArray<unsigned long long int>(host_sum);
}


TEST_F(stdgpu_cuda_atomic, float_sub)
{
    const stdgpu::index64_t N = 5000;

    // Create sequence
    float* numbers = createDeviceArray<float>(N);

    thrust::sequence(stdgpu::device_begin(numbers), stdgpu::device_end(numbers),
                     1.0f);

    float* sum = createDeviceArray<float>(1, static_cast<float>(N * (N + 1) / 2));


    thrust::for_each(stdgpu::device_begin(numbers), stdgpu::device_end(numbers),
                     subtract<float>(sum));


    float* host_sum = copyCreateDevice2HostArray(sum, 1);

    EXPECT_FLOAT_EQ(*host_sum, 0.0f);


    destroyDeviceArray<float>(numbers);
    destroyDeviceArray<float>(sum);
    destroyHostArray<float>(host_sum);
}


class random_float
{
    public:
        STDGPU_HOST_DEVICE
        random_float(const std::size_t seed,
                     const float min,
                     const float max)
            : _seed(seed),
              _min(min),
              _max(max)
        {

        }

        STDGPU_HOST_DEVICE float
        operator()(const stdgpu::index_t n) const
        {
            thrust::default_random_engine rng(static_cast<thrust::default_random_engine::result_type>(_seed));
            thrust::uniform_real_distribution<float> dist(_min, _max);
            rng.discard(n);

            return dist(rng);
        }

    private:
        std::size_t _seed;
        float _min, _max;
};


class find_min
{
    public:
        find_min(float* value)
            : _value(value)
        {

        }

        STDGPU_DEVICE_ONLY void
        operator()(const float x)
        {
            atomicMin(_value, x);
        }

    private:
        float* _value;
};


class find_max
{
    public:
        find_max(float* value)
            : _value(value)
        {

        }

        STDGPU_DEVICE_ONLY void
        operator()(const float x)
        {
            atomicMax(_value, x);
        }

    private:
        float* _value;
};


TEST_F(stdgpu_cuda_atomic, float_min)
{
    const stdgpu::index64_t N = 10000000;
    // thrust::uniform_real_distribution is not stable with std::numeric_limits<float>::{lowest(), max()}
    const float global_min = -1e38f;
    const float global_max =  1e38f;

    // Create random numbers
    float* numbers = createDeviceArray<float>(N);

    thrust::transform(thrust::counting_iterator<stdgpu::index_t>(0),
                      thrust::counting_iterator<stdgpu::index_t>(N),
                      stdgpu::device_begin(numbers),
                      random_float(test_utils::random_seed(),
                                           global_min,
                                           global_max));

    float* min = createDeviceArray<float>(1, std::numeric_limits<float>::max());


   thrust::for_each(stdgpu::device_begin(numbers), stdgpu::device_end(numbers),
                     find_min(min));


    float* host_min     = copyCreateDevice2HostArray(min,     1);
    float* host_numbers = copyCreateDevice2HostArray(numbers, N);


    bool min_found = false;
    for (stdgpu::index64_t i = 0; i < N; ++i)
    {
        // min <= numbers[i]
        EXPECT_LE(*host_min, host_numbers[i]);

        // min in numbers
        // *host_min == host_numbers[i]
        if (std::abs(*host_min - host_numbers[i]) < std::numeric_limits<float>::min())
        {
            min_found = true;
        }
    }
    EXPECT_TRUE(min_found);


    destroyDeviceArray<float>(numbers);
    destroyDeviceArray<float>(min);
    destroyHostArray<float>(host_numbers);
    destroyHostArray<float>(host_min);
}


TEST_F(stdgpu_cuda_atomic, float_max)
{
    const stdgpu::index64_t N = 10000000;
    // thrust::uniform_real_distribution is not stable with std::numeric_limits<float>::{lowest(), max()}
    const float global_min = -1e38f;
    const float global_max =  1e38f;

    // Create random numbers
    float* numbers = createDeviceArray<float>(N);

    thrust::transform(thrust::counting_iterator<stdgpu::index_t>(0),
                      thrust::counting_iterator<stdgpu::index_t>(N),
                      stdgpu::device_begin(numbers),
                      random_float(test_utils::random_seed(),
                                           global_min,
                                           global_max));

    float* max = createDeviceArray<float>(1, std::numeric_limits<float>::lowest());


    thrust::for_each(stdgpu::device_begin(numbers), stdgpu::device_end(numbers),
                     find_max(max));


    float* host_max     = copyCreateDevice2HostArray(max,     1);
    float* host_numbers = copyCreateDevice2HostArray(numbers, N);


    bool max_found = false;
    for (stdgpu::index64_t i = 0; i < N; ++i)
    {
        // max >= numbers[i]
        EXPECT_GE(*host_max, host_numbers[i]);

        // max in numbers
        // *host_max == host_numbers[i]
        if (std::abs(*host_max - host_numbers[i]) < std::numeric_limits<float>::min())
        {
            max_found = true;
        }
    }
    EXPECT_TRUE(max_found);


    destroyDeviceArray<float>(numbers);
    destroyDeviceArray<float>(max);
    destroyHostArray<float>(host_numbers);
    destroyHostArray<float>(host_max);
}


